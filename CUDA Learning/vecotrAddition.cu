#include<hip/hip_runtime.h>

#include<assert.h>
#include<cmath>

__global__ void multiply_kernel(int * A, int * B, int* C, int n) {

	int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (tid < n) {
		C[tid] = A[tid] + B[tid];
	}
}

void error_check(int * A, int* B, int* C, int n) {
	for (size_t i = 0; i < n; i++)
	{
		assert(C[i] == A[i] + B[i]);
	}
}


void arrayInit(int* array, int n) {
	for (int i = 0; i < n; i++) {
		array[i] = rand() % 10000 / 1000.0f;
	}
}

int main() {
	
	int n = 1 << 16;
	//host memory
	int* h_A = new int[n];
	int* h_B = new int[n];
	int* h_C = new int[n];

	//device memory
	int* d_A, * d_B, *d_C;

	hipMalloc(&d_A, n);
	hipMalloc(&d_B, n);
	hipMalloc(&d_C, n);
	
	arrayInit(h_A, n);
	arrayInit(h_B, n);

	hipMemcpy(d_A, h_A, sizeof(int)*n, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, sizeof(int) * n, hipMemcpyHostToDevice);

	//here put number of thread best according to the architecture of the CPU
	//but it is good to make size of multiple of 32 because these havr to translate it to warps which are of size 32
	int NUM_THREADS = 256;
	int NUM_BLOCKS = int(ceil(n / NUM_THREADS));

	//launch kernel on different stream 
	multiply_kernel<<<NUM_BLOCKS, NUM_THREADS>>>(d_A, d_B, d_C, n);

	hipMemcpy(h_C, d_C, sizeof(int) * n, hipMemcpyDeviceToHost);

	error_check(h_A, h_B, h_C, n);
	delete[] h_A;
	delete[] h_B;

	return 0;
}